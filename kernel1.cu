
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
using namespace std;
#include <iterator>
#include <algorithm>
#include <random>
#include <math.h>

// Cada thread deberia calcular la carga de 1 punto
__global__ void calcular_carga(float* iones_x, float* iones_y, float* cargas, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    
	if(tId < 8192 * 8192) {
        
        float x = tId % 8192;
        float y = tId / 8192;
        
        float carga = 0;
        float distancia;
        float x_2, y_2;
        for (int i = 0; i < cantidad; i++)  {
            x_2 = (x - iones_x[i]) * (x - iones_x[i]);
            y_2 = (y - iones_y[i]) * (y - iones_y[i]);
            distancia = sqrt(x_2 + y_2);
            carga += distancia != 0 ? 1.0 / distancia : 1;

            
        }
    
        cargas[tId] = carga;
    }
    
}

// cada thread calcula la menor carga en su fila y la guarda

__global__ void calcular_carga_fila(float* iones_x, float* iones_y, float* cargas, float*cargas_menores, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    

    if(tId < 8192) {
        float Q_menor = cargas[tId * 8192];
        float y = tId;
        float x;
        

        for (int i = tId*8192; i < tId * 8192 + 8192; i++)  {
            if(cargas[i] <Q_menor){
                Q_menor = cargas[i];
                x = i%8192;
            }
        }
        cargas_menores[tId*3] = Q_menor;
        cargas_menores[tId*3+1] = x;
        cargas_menores[tId*3+2] = y;
    }
    
}
// Calculamos entre todas la menor y ponemos la carga ahí
__global__ void posicionar_ion(float* iones_x, float* iones_y, float*cargas_menores, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    

    if(tId < 1) {
        float Q_menor = cargas_menores[0];
        float x = cargas_menores[1];
        float y = cargas_menores[2];

        for (int i = 0; i < 8192*3; i+=3)  {
            
            if(cargas_menores[i] < Q_menor){
                
                Q_menor = cargas_menores[i];
                
                x = cargas_menores[i+1];
                y = cargas_menores[i+2];
                
            }
        //    printf("%f %f %f %f\n", cargas_menores[i], Q_menor, cargas_menores[i+1], cargas_menores[i+2]);
        }
        iones_x[cantidad] = x;
        iones_y[cantidad] = y; 
    }

    
    
}



int main(int argc, char const *argv[])
{
    
    float *gpu_cargas, *cargas_menores, *gpu_iones_x, *gpu_iones_y, *iones_x, *iones_y;
    hipEvent_t ct1, ct2;
    float dt;
    int cantidad;
    iones_x = new float[6000];
    iones_y = new float[6000];

    int block_size = 256;
    int grid_size = (int) ceil( (float) 8192*8192 / block_size);
    int grid_size_b = (int) ceil( (float) 8192 / block_size);
    int grid_size_c = (int) ceil( (float) 1 / block_size);

    
    FILE *in = fopen("dataset", "r");
    for (int i = 0; i < 5000; i++)
    {
        fscanf(in, "%f %f", &iones_x[i], &iones_y[i]);
    }


    hipMalloc(&gpu_iones_x, sizeof(float) * 6000);
    hipMalloc(&gpu_iones_y, sizeof(float) * 6000);
    hipMalloc(&gpu_cargas, sizeof(float) * 8192 * 8192);
    hipMalloc(&cargas_menores, sizeof(float) * 8192*3);

    hipMemcpy(gpu_iones_x, iones_x ,sizeof(float) * 6000, hipMemcpyHostToDevice);
    hipMemcpy(gpu_iones_y, iones_y ,sizeof(float) * 6000, hipMemcpyHostToDevice);

    hipEventCreate(&ct1);
	hipEventCreate(&ct2);
    hipEventRecord(ct1);

    
    for (cantidad = 5000; cantidad < 5010; cantidad++)
    {
        
        
        calcular_carga<<<grid_size, block_size>>>(gpu_iones_x, gpu_iones_y, gpu_cargas, cantidad);
        hipDeviceSynchronize();

        calcular_carga_fila<<<grid_size_b, block_size>>>(gpu_iones_x, gpu_iones_y, gpu_cargas, cargas_menores, cantidad);
        hipDeviceSynchronize();

        posicionar_ion<<<grid_size_c, block_size>>>(gpu_iones_x, gpu_iones_y, cargas_menores, cantidad);
        hipDeviceSynchronize();
        hipMemcpy(iones_x, gpu_iones_x,sizeof(float) * 6000, hipMemcpyDeviceToHost);
        hipMemcpy(iones_y, gpu_iones_y,sizeof(float) * 6000, hipMemcpyDeviceToHost);
        cout << iones_x[cantidad] << " " << iones_y[cantidad] << endl;
    }

    hipEventRecord(ct2);
	hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);

    cout << "Tiempo: " << dt << "[ms]" << '\n';
    
    hipFree(gpu_iones_x);
    hipFree(gpu_iones_y);
    hipFree(gpu_cargas);
    hipFree(cargas_menores);
    hipFree(gpu_iones_x);
    hipFree(gpu_iones_y);

    delete iones_x;
    delete iones_y;
    
    return 0;
}